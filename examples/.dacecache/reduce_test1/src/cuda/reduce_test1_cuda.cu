#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <dace/dace.h>


struct reduce_test1_t {
    dace::cuda::Context *gpu_context;
};



DACE_EXPORTED int __dace_init_cuda(reduce_test1_t *__state);
DACE_EXPORTED void __dace_exit_cuda(reduce_test1_t *__state);

DACE_DFI void reduce_test1_5_4_12_8_0_0_13(double& __tmp_13_37_r, double * __tmp_15_16_w, long long threadIdx_x) {
    double reduced[1]  DACE_ALIGN(64);

    {

        {
            double __a = __tmp_13_37_r;
            double __out;

            ///////////////////

            __out = dace::warpReduce<dace::ReductionType::Sum, double>::reduce(__a);

            ///////////////////

            reduced[0] = __out;
        }

    }
    if ((threadIdx_x == 0)) {
        {

            {
                double __inp = reduced[0];
                double __out;

                ///////////////////
                // Tasklet code (assign_15_16)
                __out = __inp;
                ///////////////////

                dace::wcr_fixed<dace::ReductionType::Sum, double>::reduce_atomic(__tmp_15_16_w, __out);
            }

        }
        goto __state_1_endif_14;

    }
    __state_1_endif_14:;
    
}



int __dace_init_cuda(reduce_test1_t *__state) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    hipMalloc((void **) &dev_X, 1);
    hipFree(dev_X);

    __state->gpu_context = new dace::cuda::Context(2, 1);

    // Create cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamCreateWithFlags(&__state->gpu_context->streams[i], hipStreamNonBlocking);
    }
    for(int i = 0; i < 1; ++i) {
        hipEventCreateWithFlags(&__state->gpu_context->events[i], hipEventDisableTiming);
    }

    

    return 0;
}

void __dace_exit_cuda(reduce_test1_t *__state) {
    

    // Destroy cuda streams and events
    for(int i = 0; i < 2; ++i) {
        hipStreamDestroy(__state->gpu_context->streams[i]);
    }
    for(int i = 0; i < 1; ++i) {
        hipEventDestroy(__state->gpu_context->events[i]);
    }

    delete __state->gpu_context;
}

__global__ void reduce_test1_5_0_0_6(double * __restrict__ gpu___return, const double * __restrict__ gpu_inputs) {
    {
        {
            int blockIdx_x = blockIdx.x;
            int blockIdx_y = blockIdx.y;
            __shared__ double shared[1024];
            {
                {
                    {
                        double value;
                        int threadIdx_x = threadIdx.x;
                        int threadIdx_y = threadIdx.y;
                        if (threadIdx_x < 32) {
                            if (threadIdx_y < 32) {
                                {
                                    for (auto i = 0; i < 2; i += 1) {
                                        {
                                            double __inp = gpu_inputs[(((((32 * blockIdx_x) + (4096 * blockIdx_y)) + (8192 * i)) + threadIdx_x) + (128 * threadIdx_y))];
                                            double __out;

                                            ///////////////////
                                            // Tasklet code (assign_10_16)
                                            __out = __inp;
                                            ///////////////////

                                            dace::wcr_fixed<dace::ReductionType::Sum, double>::reduce(&value, __out);
                                        }
                                    }
                                }
                                {
                                    double __inp = value;
                                    double __out;

                                    ///////////////////
                                    // Tasklet code (assign_11_12)
                                    __out = __inp;
                                    ///////////////////

                                    shared[((32 * threadIdx_x) + threadIdx_y)] = __out;
                                }
                                {
                                    double __out;

                                    ///////////////////
                                    // Tasklet code (_convert_to_float64_)
                                    __out = dace::float64(0);
                                    ///////////////////

                                    value = __out;
                                }
                            }
                        }
                    }
                }
            }
            __syncthreads();
            {
                {
                    {
                        int threadIdx_x = threadIdx.x;
                        int threadIdx_y = threadIdx.y;
                        if (threadIdx_x < 32) {
                            if (threadIdx_y < 32) {
                                reduce_test1_5_4_12_8_0_0_13(shared[(threadIdx_x + (32 * threadIdx_y))], &gpu___return[((32 * blockIdx_x) + threadIdx_y)], threadIdx_x);
                            }
                        }
                    }
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_reduce_test1_5_0_0_6(reduce_test1_t *__state, double * __restrict__ gpu___return, const double * __restrict__ gpu_inputs);
void __dace_runkernel_reduce_test1_5_0_0_6(reduce_test1_t *__state, double * __restrict__ gpu___return, const double * __restrict__ gpu_inputs)
{

    void  *reduce_test1_5_0_0_6_args[] = { (void *)&gpu___return, (void *)&gpu_inputs };
    hipLaunchKernel((void*)reduce_test1_5_0_0_6, dim3(int_ceil(4, 1), int_ceil(2, 1), 1), dim3(32, 32, 1), reduce_test1_5_0_0_6_args, 0, __state->gpu_context->streams[1]);
}
__global__ void assign_4_4_map_0_0_2(double * __restrict__ gpu___return) {
    {
        int __i0 = (blockIdx.x * 32 + threadIdx.x);
        if (__i0 < 128) {
            {
                double __out;

                ///////////////////
                // Tasklet code (assign_4_4)
                __out = 0;
                ///////////////////

                gpu___return[__i0] = __out;
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_assign_4_4_map_0_0_2(reduce_test1_t *__state, double * __restrict__ gpu___return);
void __dace_runkernel_assign_4_4_map_0_0_2(reduce_test1_t *__state, double * __restrict__ gpu___return)
{

    void  *assign_4_4_map_0_0_2_args[] = { (void *)&gpu___return };
    hipLaunchKernel((void*)assign_4_4_map_0_0_2, dim3(int_ceil(int_ceil(128, 1), 32), int_ceil(1, 1), int_ceil(1, 1)), dim3(32, 1, 1), assign_4_4_map_0_0_2_args, 0, __state->gpu_context->streams[0]);
}

